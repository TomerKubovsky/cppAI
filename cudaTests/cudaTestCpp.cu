#include <iostream>
#include <hip/hip_runtime.h>

#include <chrono>
#include <thread>

void fillArr(int* arr, int valTF, int size)
{
	for (int i = 0; i < size; i++)
	{
		arr[i] = valTF;
	}
}


__global__ void addFunc(const int* a1, const int* a2, int* a3, const unsigned long long arrSize)
{

	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// const unsigned int stride = blockDim.x * gridDim.x;

	a3[idx] = a1[threadIdx.x] + a2[threadIdx.x];

	// for (int i = idx; i < arrSize; i += stride)
	// {
		// a3[i] = a1[i] + a2[i];
	// }
}

void cpuAdd(const int* a1, const int* a2, int* a3, const unsigned long long arrSize)
{
	for (int i = 0; i < arrSize; i++)
	{
		a3[i] = a1[i] + a2[i];
	}
}

int main()
{
	const unsigned long long N = 2<<20;
	int *DA1, *DA2, *DA3, *A1(new int[N]), *A2(new int[N]), *A3(new int[N]), *HOA3(new int[N]);


	fillArr(A1, 3, N);
	fillArr(A2, 3, N);

	hipMalloc((void**)&DA1, N * sizeof(int));
	hipMalloc((void**)&DA2, N * sizeof(int));
	hipMalloc((void**)&DA3, N * sizeof(int));

	hipMemcpy(DA1, A1, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(DA2, A2, N * sizeof(int), hipMemcpyHostToDevice);

	constexpr unsigned int blockSize = 256;

	constexpr unsigned int blockCount = (N + (blockSize - 1)) / blockSize;

	auto start = std::chrono::high_resolution_clock::now();

	hipEvent_t DStart, stop;
	float milliseconds = 0.0f;
	hipEventCreate(&DStart);
	hipEventCreate(&stop);

	hipEventRecord(DStart);

	addFunc <<< blockCount, blockSize >>> (DA1, DA2, DA3, N);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, DStart, stop);
	std::cout << "GPU Task Time: " << milliseconds << std::endl;
	hipDeviceSynchronize();

	hipEventDestroy(DStart);
	hipEventDestroy(stop);


	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> elapsed = end - start;

	// std::cout << "GPU Task Time: " << elapsed.count() << " ms\n";

	hipMemcpy(HOA3, DA3, N * sizeof(int), hipMemcpyDeviceToHost);

	start = std::chrono::high_resolution_clock::now();
	cpuAdd(A1, A2, A3, N);
	end = std::chrono::high_resolution_clock::now();
	elapsed = end - start;

	std::cout << "CPU Task Time: " << elapsed.count() << " ms\n";

	printf("o1: %i, o10: %i, o100: %i, o10000: %i", HOA3[0], HOA3[10], HOA3[100], HOA3[10000]);

	delete[] A1;
	delete[] A2;
	delete[] A3;
	delete[] HOA3;

	hipFree(DA1);
	hipFree(DA2);
	hipFree(DA3);
}


